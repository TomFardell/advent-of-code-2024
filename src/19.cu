
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define P

#ifdef P
#define FILE_NAME "data/19p.txt"
#define TC 447
#define PC 400
#define TL 10
#define PL 62
#endif
#ifdef S
#define FILE_NAME "data/19s.txt"
#define TC 8
#define PC 8
#define TL 5
#define PL 8
#endif

// Number of makeable and unmakeable patterns to store. Setting this too large slows the program,
// since there will be redundant values stored. I am using an array rather than a hash table, so the
// lookup time is proportional to this value
#define MC 100
#define LT (TL - 2)  // Longest towel is 2 less than the size of buffer needed

typedef struct {
  char makeables[MC * PL], unmakeables[MC * PL];
  int n_m, n_u;
  int c_m, c_u;
} storage;

int can_be_made(const char *pattern, const char *towels, storage *p_store) {
  int p_len = strlen(pattern);
  if (p_len == 0) return 1;

  // For each size up to the minimum of this pattern's length and the maximum towel length
  for (int i = 0; i < ((p_len < LT) ? p_len : LT); i++) {
    // Get the prefix of pattern of this length
    char prefix[TL];
    strncpy(prefix, pattern, i + 1);
    prefix[i + 1] = 0;

    for (int t = 0; t < TC; t++) {
      // If the prefix matches a towel
      if (!strcmp(towels + TL * t, prefix)) {
        const char *suffix = pattern + i + 1;
        int makeable = -1;
        int new_suffix = 1;

        for (int m = 0; m < p_store->n_m && makeable == -1; m++) {
          if (!strcmp(suffix, p_store->makeables + PL * m)) {
            makeable = 1;
          }
        }

        // p_store->unmakeables is likely to contain null strings, so first ensure the suffix is not
        // null (as otherwise it will be seen as unmakeable)
        if (strlen(suffix) > 0) {
          for (int u = 0; u < p_store->n_u && makeable == -1; u++) {
            if (!strcmp(suffix, p_store->unmakeables + PL * u)) {
              makeable = 0;
              new_suffix = 0;
            }
          }
        }

        // Only recurse if not already seen the suffix
        if (makeable == -1) makeable = can_be_made(suffix, towels, p_store);

        if (makeable) {
          // Put the pattern in the array of makeables, wrapping the count if the array is full
          p_store->c_m = (p_store->c_m + 1) % MC;
          p_store->n_m = (p_store->n_m + 1 > MC) ? MC : p_store->n_m + 1;
          strncpy(p_store->makeables + PL * p_store->c_m, pattern, PL);
          return 1;
        } else if (new_suffix) {
          // Put the suffix in the array of unmakeables, wrapping the count if the array is full
          p_store->c_u = (p_store->c_u + 1) % MC;
          p_store->n_u = (p_store->n_u + 1 > MC) ? MC : p_store->n_u + 1;
          strncpy(p_store->unmakeables + PL * p_store->c_u, suffix, PL);
        }
      }
    }
  }

  return 0;
}

int main() {
  char towels[TC * TL];
  char patterns[PC * PL];

  FILE *file = fopen(FILE_NAME, "r");
  for (int i = 0; i < TC - 1; i++) {
    fscanf(file, "%[^,]s", towels + TL * i);
    fscanf(file, ", ");
  }
  fscanf(file, "%s", towels + TL * (TC - 1));
  fscanf(file, "\n\n");
  for (int i = 0; i < PC; i++) {
    fscanf(file, "%s ", patterns + PL * i);
  }
  fclose(file);

  storage p_store = {{0}, {0}, 0, 0, 0, 0};

  int count = 0;
  for (int p = 0; p < PC; p++) {
    int m = can_be_made(patterns + PL * p, towels, &p_store);
    count += m;
  }

  printf("%d\n", count);

  return 0;
}
