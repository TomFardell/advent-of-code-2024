#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "utils.cuh"

void process_cuda_error(hipError_t err) {
  if (err != hipSuccess) {
    fprintf(stderr, "GPU error\n%d %s: %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}